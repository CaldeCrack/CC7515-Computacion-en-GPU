
#include <hip/hip_runtime.h>
#include <cassert>

typedef unsigned char ubyte;

__global__ void fillRandomLifeData(ubyte *lifeData, size_t size,
                                   unsigned int seed) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (size_t i = idx; i < size; i += blockDim.x * gridDim.x) {
    unsigned int x = i ^ seed;
    x = (x * 1664525u + 1013904223u);
    lifeData[i] = (x >> 24) & 1;
  }
}

__global__ void simpleLifeKernel(volatile const ubyte *lifeData,
                                 uint worldWidth, uint worldHeight,
                                 ubyte *resultLifeData) {
  uint worldSize = worldWidth * worldHeight;

  for (uint cellId = blockIdx.x * blockDim.x + threadIdx.x; cellId < worldSize;
       cellId += blockDim.x * gridDim.x) {

    uint x = cellId % worldWidth;
    uint yAbs = cellId - x;

    uint xLeft = (x + worldWidth - 1) % worldWidth;
    uint xRight = (x + 1) % worldWidth;

    uint yAbsUp = (yAbs + worldSize - worldWidth) % worldSize;
    uint yAbsDown = (yAbs + worldWidth) % worldSize;

    uint aliveCells = lifeData[xLeft + yAbsUp] + lifeData[x + yAbsUp] +
                      lifeData[xRight + yAbsUp] + lifeData[xLeft + yAbs] +
                      lifeData[xRight + yAbs] + lifeData[xLeft + yAbsDown] +
                      lifeData[x + yAbsDown] + lifeData[xRight + yAbsDown];

    resultLifeData[x + yAbs] =
        aliveCells == 3 || (aliveCells == 2 && lifeData[x + yAbs]) ? 1 : 0;
  }
}

__global__ void simpleLifeKernelIfs(volatile const ubyte *lifeData,
                                    uint worldWidth, uint worldHeight,
                                    ubyte *resultLifeData) {
  uint worldSize = worldWidth * worldHeight;

  for (uint cellId = blockIdx.x * blockDim.x + threadIdx.x; cellId < worldSize;
       cellId += blockDim.x * gridDim.x) {

    uint x = cellId % worldWidth;
    uint yAbs = cellId - x;

    uint xLeft = (x + worldWidth - 1) % worldWidth;
    uint xRight = (x + 1) % worldWidth;

    uint yAbsUp = (yAbs + worldSize - worldWidth) % worldSize;
    uint yAbsDown = (yAbs + worldWidth) % worldSize;

    uint aliveCells = 0;
    if (lifeData[xLeft + yAbsUp])
      aliveCells += 1;
    if (lifeData[x + yAbsUp])
      aliveCells += 1;
    if (lifeData[xRight + yAbsUp])
      aliveCells += 1;
    if (lifeData[xLeft + yAbs])
      aliveCells += 1;
    if (lifeData[xRight + yAbs])
      aliveCells += 1;
    if (lifeData[xLeft + yAbsDown])
      aliveCells += 1;
    if (lifeData[x + yAbsDown])
      aliveCells += 1;
    if (lifeData[xRight + yAbsDown])
      aliveCells += 1;

    if (aliveCells == 3 || (aliveCells == 2 && lifeData[x + yAbs]))
      resultLifeData[x + yAbs] = 1;
    else
      resultLifeData[x + yAbs] = 0;
  }
}

__global__ void simpleLifeKernel2D(volatile ubyte *const *lifeData,
                                   uint worldWidth, uint worldHeight,
                                   ubyte **resultLifeData) {
  uint x = blockIdx.x * blockDim.x + threadIdx.x;
  uint y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= worldWidth || y >= worldHeight)
    return;

  uint xLeft = (x + worldWidth - 1) % worldWidth;
  uint xRight = (x + 1) % worldWidth;
  uint yUp = (y + worldHeight - 1) % worldHeight;
  uint yDown = (y + 1) % worldHeight;

  uint aliveCells = lifeData[yUp][xLeft] + lifeData[yUp][x] +
                    lifeData[yUp][xRight] + lifeData[y][xLeft] +
                    lifeData[y][xRight] + lifeData[yDown][xLeft] +
                    lifeData[yDown][x] + lifeData[yDown][xRight];

  resultLifeData[y][x] =
      (aliveCells == 3 || (aliveCells == 2 && lifeData[y][x])) ? 1 : 0;
}

void runSimpleLifeKernel(ubyte *&d_lifeData, ubyte *&d_lifeDataBuffer,
                         size_t worldWidth, size_t worldHeight,
                         size_t iterationsCount, ushort threadsCount) {
  size_t reqBlocksCount = (worldWidth * worldHeight) / threadsCount;
  ushort blocksCount = (ushort)std::min((size_t)32768, reqBlocksCount);

  for (size_t i = 0; i < iterationsCount; ++i) {
    simpleLifeKernel<<<blocksCount, threadsCount>>>(
        d_lifeData, worldWidth, worldHeight, d_lifeDataBuffer);
    std::swap(d_lifeData, d_lifeDataBuffer);
  }
  hipDeviceSynchronize();
}

void runSimpleLifeKernelIfs(ubyte *&d_lifeData, ubyte *&d_lifeDataBuffer,
                            size_t worldWidth, size_t worldHeight,
                            size_t iterationsCount, ushort threadsCount) {
  size_t reqBlocksCount = (worldWidth * worldHeight) / threadsCount;
  ushort blocksCount = (ushort)std::min((size_t)32768, reqBlocksCount);

  for (size_t i = 0; i < iterationsCount; ++i) {
    simpleLifeKernelIfs<<<blocksCount, threadsCount>>>(
        d_lifeData, worldWidth, worldHeight, d_lifeDataBuffer);
    std::swap(d_lifeData, d_lifeDataBuffer);
  }
  hipDeviceSynchronize();
}

void runSimpleLifeKernel2D(ubyte **&d_lifeData, ubyte **&d_lifeDataBuffer,
                           size_t worldWidth, size_t worldHeight,
                           size_t iterationsCount, ushort threadsCount) {
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((worldWidth + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (worldHeight + threadsPerBlock.y - 1) / threadsPerBlock.y);

  for (size_t i = 0; i < iterationsCount; ++i) {
    simpleLifeKernel2D<<<numBlocks, threadsPerBlock>>>(
        d_lifeData, worldWidth, worldHeight, d_lifeDataBuffer);
    std::swap(d_lifeData, d_lifeDataBuffer);
  }
  hipDeviceSynchronize();
}
