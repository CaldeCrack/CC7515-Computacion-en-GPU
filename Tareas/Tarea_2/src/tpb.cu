
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <chrono>
#include <fstream>
#include <iostream>
#include <vector>

typedef unsigned char ubyte;
typedef unsigned short ushort;

void runSimpleLifeKernel(ubyte *&d_lifeData, ubyte *&d_lifeDataBuffer,
                         size_t worldWidth, size_t worldHeight,
                         size_t iterationsCount, ushort threadsCount);

void runSimpleLifeKernelIfs(ubyte *&d_lifeData, ubyte *&d_lifeDataBuffer,
                            size_t worldWidth, size_t worldHeight,
                            size_t iterationsCount, ushort threadsCount);

void runSimpleLifeKernel2D(ubyte **&d_lifeData, ubyte **&d_lifeDataBuffer,
                           size_t worldWidth, size_t worldHeight,
                           size_t iterationsCount, ushort threadsCount);

__global__ void fillRandomLifeData(ubyte *lifeData, size_t size,
                                   unsigned int seed);

void runExperiment1D(int iterations, ushort threads, size_t height,
                     size_t width, std::ofstream &outfile, std::string title) {
  size_t totalCells = height * width;
  ubyte *d_lifeData = nullptr, *d_lifeDataBuffer = nullptr;

  hipMalloc(&d_lifeData, totalCells * sizeof(ubyte));
  hipMalloc(&d_lifeDataBuffer, totalCells * sizeof(ubyte));

  unsigned int seed = static_cast<unsigned int>(time(nullptr));
  int blocks = std::min((totalCells + threads - 1) / threads, 32768UL);
  std::vector<double> timings;

  for (ushort i = 0; i < 15; ++i) {
    hipMemcpy(d_lifeDataBuffer, d_lifeData, totalCells * sizeof(ubyte),
               hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();

    fillRandomLifeData<<<blocks, threads>>>(d_lifeData, totalCells, seed++);
    hipDeviceSynchronize();

    double duration;
    if (title == "CUDA") { // xd
      auto start = std::chrono::high_resolution_clock::now();
      runSimpleLifeKernel(d_lifeData, d_lifeDataBuffer, width, height,
                          iterations, threads);
      auto end = std::chrono::high_resolution_clock::now();

      duration =
          std::chrono::duration_cast<std::chrono::microseconds>(end - start)
              .count();
    } else {
      auto start = std::chrono::high_resolution_clock::now();
      runSimpleLifeKernelIfs(d_lifeData, d_lifeDataBuffer, width, height,
                             iterations, threads);
      auto end = std::chrono::high_resolution_clock::now();

      duration =
          std::chrono::duration_cast<std::chrono::microseconds>(end - start)
              .count();
    }

    timings.push_back(duration);
  }

  std::sort(timings.begin(), timings.end());
  double medianTime = timings[timings.size() / 2];
  double cellsPerSecond = (double)(totalCells * iterations) / medianTime * 1e6;

  outfile << title << ',' << width << ',' << height << ',' << totalCells << ','
          << threads << ',' << iterations << ',' << medianTime << ','
          << cellsPerSecond << '\n';

  hipFree(d_lifeData);
  hipFree(d_lifeDataBuffer);
}

void runExperiment2D(int iterations, ushort threads, size_t height,
                     size_t width, std::ofstream &outfile, std::string title) {
  size_t totalCells = height * width;

  ubyte **d_lifeData = nullptr;
  ubyte **d_lifeDataBuffer = nullptr;
  ubyte *d_lifeDataRows = nullptr;
  ubyte *d_lifeDataBufferRows = nullptr;

  hipMalloc(&d_lifeDataRows, totalCells * sizeof(ubyte));
  hipMalloc(&d_lifeDataBufferRows, totalCells * sizeof(ubyte));
  hipMalloc(&d_lifeData, height * sizeof(ubyte *));
  hipMalloc(&d_lifeDataBuffer, height * sizeof(ubyte *));

  std::vector<ubyte *> h_lifeData(height);
  std::vector<ubyte *> h_lifeDataBuffer(height);
  for (size_t i = 0; i < height; ++i) {
    h_lifeData[i] = d_lifeDataRows + i * width;
    h_lifeDataBuffer[i] = d_lifeDataBufferRows + i * width;
  }

  hipMemcpy(d_lifeData, h_lifeData.data(), height * sizeof(ubyte *),
             hipMemcpyHostToDevice);
  hipMemcpy(d_lifeDataBuffer, h_lifeDataBuffer.data(),
             height * sizeof(ubyte *), hipMemcpyHostToDevice);

  unsigned int seed = static_cast<unsigned int>(time(nullptr));
  int blocks = std::min((totalCells + threads - 1) / threads, 32768UL);
  std::vector<double> timings;

  for (int i = 0; i < 31; ++i) {
    hipMemcpy(d_lifeDataBufferRows, d_lifeDataRows, totalCells * sizeof(ubyte),
               hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();

    fillRandomLifeData<<<blocks, threads>>>(d_lifeDataRows, totalCells, seed++);
    hipDeviceSynchronize();

    auto start = std::chrono::high_resolution_clock::now();
    runSimpleLifeKernel2D(d_lifeData, d_lifeDataBuffer, width, height,
                          iterations, threads);
    auto end = std::chrono::high_resolution_clock::now();

    double duration =
        std::chrono::duration_cast<std::chrono::microseconds>(end - start)
            .count();
    timings.push_back(duration);
  }

  std::sort(timings.begin(), timings.end());
  double medianTime = timings[timings.size() / 2];
  double cellsPerSecond = (double)(totalCells * iterations) / medianTime * 1e6;

  outfile << title << ',' << width << ',' << height << ',' << totalCells << ','
          << threads << ',' << iterations << ',' << medianTime << ','
          << cellsPerSecond << '\n';

  hipFree(d_lifeData);
  hipFree(d_lifeDataBuffer);
  hipFree(d_lifeDataRows);
  hipFree(d_lifeDataBufferRows);
}

void experiment(int iterations, ushort threads, size_t height, size_t width,
                std::ofstream &outfile) {
  // Optimal case
  runExperiment1D(iterations, threads, height, width, outfile, "CUDA");

  // Ifs case
  runExperiment1D(iterations, threads, height, width, outfile, "CUDA Ifs");

  // 2D case
  runExperiment2D(iterations, threads, height, width, outfile, "CUDA 2D");
}

int main() {
  srand(static_cast<unsigned>(time(nullptr)));

  const int iterations = 16;
  const ushort threadOptions[5] = {64, 128, 256, 512, 1024};
  std::ofstream outfile("cuda_benchmark.csv");
  outfile << "Mode,Width,Height,Length,Threads,Iterations,Time[μs],"
             "Cells/s\n";

  size_t worldWidth = 1ull << 16;

  for (ushort exp = 4; exp <= 16; ++exp) {
    size_t worldHeight = 1ull << exp;

    for (ushort threads : threadOptions)
      experiment(iterations, threads, worldHeight, worldWidth, outfile);
  }

  outfile.close();
  return 0;
}
